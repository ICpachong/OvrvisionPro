#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/core/core.hpp>

using namespace cv;
#ifdef OPENCV_VERSION_2_4
#include <opencv2/gpu/gpu.hpp>
using namespace cv::gpu;
#else
#include <opencv2/core/cuda.hpp>
using namespace cv::cuda;
#endif
#include <opencv2/imgproc/imgproc.hpp>


namespace OVR
{
	namespace CUDA
	{
		struct Bayer2BGR
		{
			ushort patch[4][4];
			__device__ void shr_8()
			{
				patch[0][0] >>= 8;
				patch[0][1] >>= 8;
				patch[0][2] >>= 8;
				patch[0][3] >>= 8;
				patch[1][0] >>= 8;
				patch[1][1] >>= 8;
				patch[1][2] >>= 8;
				patch[1][3] >>= 8;
				patch[2][0] >>= 8;
				patch[2][1] >>= 8;
				patch[2][2] >>= 8;
				patch[2][3] >>= 8;
				patch[3][0] >>= 8;
				patch[3][1] >>= 8;
				patch[3][2] >>= 8;
				patch[3][3] >>= 8;
			}

			__device__ void apply(int x, int y, PtrStepSz<uchar3> dst)
			{
				uchar3 bgr[2][2];
				ushort p[4][4];
				p[0][0] = patch[0][0] & 0xff;
				p[0][1] = patch[0][1] & 0xff;
				p[0][2] = patch[0][2] & 0xff;
				p[0][3] = patch[0][3] & 0xff;
				p[1][0] = patch[1][0] & 0xff;
				p[1][1] = patch[1][1] & 0xff;
				p[1][2] = patch[1][2] & 0xff;
				p[1][3] = patch[1][3] & 0xff;
				p[2][0] = patch[2][0] & 0xff;
				p[2][1] = patch[2][1] & 0xff;
				p[2][2] = patch[2][2] & 0xff;
				p[2][3] = patch[2][3] & 0xff;
				p[3][0] = patch[3][0] & 0xff;
				p[3][1] = patch[3][1] & 0xff;
				p[3][2] = patch[3][2] & 0xff;
				p[3][3] = patch[3][3] & 0xff;

				bgr[0][0].x = (uchar)((p[0][1] + p[2][1]) >> 1); // B 
				bgr[0][0].y = (uchar)(p[1][1]); // G
				bgr[0][0].z = (uchar)((p[1][0] + p[1][2]) >> 1); // R

				bgr[0][1].x = (uchar)(p[2][1]); // B
				bgr[0][1].y = (uchar)((p[2][0] + p[2][2] + p[1][1] + p[3][1]) >> 2); // G
				bgr[0][1].z = (uchar)((p[1][0] + p[3][0] + p[1][2] + p[3][2]) >> 2); // R

				bgr[1][0].x = (uchar)((p[0][1] + p[2][1] + p[0][3] + p[2][3]) >> 2); // B
				bgr[1][0].y = (uchar)((p[0][2] + p[2][2] + p[1][1] + p[1][3]) >> 2); // G
				bgr[1][0].z = (uchar)(p[1][2]); // R
					
				bgr[1][1].x = (uchar)((p[2][1] + p[2][3]) >> 1); // B 
				bgr[1][1].y = (uchar)(p[2][2]); // G
				bgr[1][1].z = (uchar)((p[1][2] + p[3][2]) >> 1); // R

				// store result
				((uchar3 *)dst.ptr(y))[x] = bgr[0][0];
				((uchar3 *)dst.ptr(y))[x + 1] = bgr[1][0];
				((uchar3 *)dst.ptr(y + 1))[x] = bgr[0][1];
				((uchar3 *)dst.ptr(y + 1))[x + 1] = bgr[1][1];
			}

			// First row of image
			__device__ void applyUpper(int x, int y, PtrStepSz<uchar3> dst)
			{
				uchar3 bgr[2][2];
				ushort p[4][4];
				p[0][0] = patch[0][0] & 0xff;
				p[0][1] = patch[0][1] & 0xff;
				p[0][2] = patch[0][2] & 0xff;
				p[0][3] = patch[0][3] & 0xff;
				p[1][0] = patch[1][0] & 0xff;
				p[1][1] = patch[1][1] & 0xff;
				p[1][2] = patch[1][2] & 0xff;
				p[1][3] = patch[1][3] & 0xff;
				p[2][0] = patch[2][0] & 0xff;
				p[2][1] = patch[2][1] & 0xff;
				p[2][2] = patch[2][2] & 0xff;
				p[2][3] = patch[2][3] & 0xff;
				p[3][0] = patch[3][0] & 0xff;
				p[3][1] = patch[3][1] & 0xff;
				p[3][2] = patch[3][2] & 0xff;
				p[3][3] = patch[3][3] & 0xff;

				bgr[0][0].x = (uchar)((p[0][1] + p[2][1]) >> 1); // B 
				bgr[0][0].y = (uchar)(p[1][1]); // G
				bgr[0][0].z = (uchar)((p[1][0] + p[1][2]) >> 1); // R

				bgr[0][1].x = (uchar)(p[2][1]); // B
				bgr[0][1].y = (uchar)((p[2][0] + p[2][2] + p[1][1] + p[3][1]) >> 2); // G
				bgr[0][1].z = (uchar)((p[1][0] + p[3][0] + p[1][2] + p[3][2]) >> 2); // R

				bgr[1][0].x = (uchar)((p[0][1] + p[2][1] + p[0][3] + p[2][3]) >> 2); // B
				bgr[1][0].y = (uchar)((p[0][2] + p[2][2] + p[1][1] + p[1][3]) >> 2); // G
				bgr[1][0].z = (uchar)(p[1][2]); // R
					
				bgr[1][1].x = (uchar)((p[2][1] + p[2][3]) >> 1); // B 
				bgr[1][1].y = (uchar)(p[2][2]); // G
				bgr[1][1].z = (uchar)((p[1][2] + p[3][2]) >> 1); // R

				// store result
				((uchar3 *)dst.ptr(y))[x] = bgr[0][0];
				((uchar3 *)dst.ptr(y))[x + 1] = bgr[1][0];
				((uchar3 *)dst.ptr(y + 1))[x] = bgr[0][1];
				((uchar3 *)dst.ptr(y + 1))[x + 1] = bgr[1][1];
			}
		};

		// Stereo bayer to RGB conversion
		// each thread calculates pixels of interest
		//
		// G|RG|R
		// ------
		// B|GB|G GB -- pixels of interest
		// G|RG|R RG -- pixels of interest
		// ------
		// B|GB|G
		// 
		__global__ void bayer2BGR(const PtrStepSz<ushort> src, PtrStepSz<uchar3> left, PtrStepSz<uchar3> right)
		{
			int s_x = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x);
			int s_y = 2 * ((blockIdx.y * blockDim.y) + threadIdx.y);

			// pixels of interest
			Bayer2BGR bayer;

			if (0 < s_y && s_y < src.rows - 2 && 0 < s_x && s_x < src.cols - 2)
			{
				bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
				//
				bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
				//
				bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				//
				bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (0 == s_y) // first row
			{
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				//
				if (s_x == 0)
				{
					bayer.patch[1][0] = bayer.patch[1][2];
					bayer.patch[2][0] = bayer.patch[2][2];
					bayer.patch[3][0] = bayer.patch[3][2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
					bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];
				} 
				else if (s_x == src.cols)
				{
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
					bayer.patch[1][3] = bayer.patch[1][1];
					bayer.patch[2][3] = bayer.patch[2][1];
					bayer.patch[3][3] = bayer.patch[3][1];
				}
				else 
				{
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
					bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];
				}

				bayer.patch[0][0] = bayer.patch[2][0];
				bayer.patch[0][1] = bayer.patch[2][1];
				bayer.patch[0][2] = bayer.patch[2][2];
				bayer.patch[0][3] = bayer.patch[2][3];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (s_y == src.rows - 2) // last row
			{
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				//
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				if (s_x == 0)
				{
					bayer.patch[0][0] = bayer.patch[0][2];
					bayer.patch[1][0] = bayer.patch[1][2];
					bayer.patch[2][0] = bayer.patch[2][2];
					bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				} 
				else if (s_x == src.cols)
				{
					bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[0][3] = bayer.patch[0][1];
					bayer.patch[1][3] = bayer.patch[1][1];
					bayer.patch[2][3] = bayer.patch[2][1];
				}
				else 
				{
					bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
					bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
					bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
					bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
					bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
					bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				}

				bayer.patch[3][0] = bayer.patch[1][0];
				bayer.patch[3][1] = bayer.patch[1][1];
				bayer.patch[3][2] = bayer.patch[1][2];
				bayer.patch[3][3] = bayer.patch[1][3];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else if (0 == s_x) // first col
			{
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				bayer.patch[0][3] = src.ptr(s_y - 1)[s_x + 2];
				//
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				bayer.patch[1][3] = src.ptr(s_y)[s_x + 2];
				//
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				bayer.patch[2][3] = src.ptr(s_y + 1)[s_x + 2];
				//
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				bayer.patch[3][3] = src.ptr(s_y + 2)[s_x + 2];

				bayer.patch[0][0] = bayer.patch[0][2];
				bayer.patch[1][0] = bayer.patch[1][2];
				bayer.patch[2][0] = bayer.patch[2][2];
				bayer.patch[3][0] = bayer.patch[3][2];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
			else // last col
			{
				bayer.patch[0][0] = src.ptr(s_y - 1)[s_x - 1];
				bayer.patch[0][1] = src.ptr(s_y - 1)[s_x];
				bayer.patch[0][2] = src.ptr(s_y - 1)[s_x + 1];
				//
				bayer.patch[1][0] = src.ptr(s_y)[s_x - 1];
				bayer.patch[1][1] = src.ptr(s_y)[s_x];
				bayer.patch[1][2] = src.ptr(s_y)[s_x + 1];
				//
				bayer.patch[2][0] = src.ptr(s_y + 1)[s_x - 1];
				bayer.patch[2][1] = src.ptr(s_y + 1)[s_x];
				bayer.patch[2][2] = src.ptr(s_y + 1)[s_x + 1];
				//
				bayer.patch[3][0] = src.ptr(s_y + 2)[s_x - 1];
				bayer.patch[3][1] = src.ptr(s_y + 2)[s_x];
				bayer.patch[3][2] = src.ptr(s_y + 2)[s_x + 1];
				//
				bayer.patch[0][3] = bayer.patch[0][1];
				bayer.patch[1][3] = bayer.patch[1][1];
				bayer.patch[2][3] = bayer.patch[2][1];
				bayer.patch[3][3] = bayer.patch[3][1];

				bayer.apply(s_x, s_y, left); // Lower bytes for Left
				bayer.shr_8();
				bayer.apply(s_x, s_y, right); // Higher bytes for Right
			}
		}

		double bayerGB2BGR(GpuMat src, GpuMat left, GpuMat right)
		{
			dim3 threads(16, 16);
			dim3 grid((src.cols / 2) / (threads.x), (src.rows / 2) / (threads.y));
			hipDeviceSynchronize();
			int64 start = getTickCount();
			bayer2BGR<<<grid, threads>>>(src, left, right);
			int64 stop = getTickCount();
			hipDeviceSynchronize();
			return (stop - start) * 1000000 / getTickFrequency();
			//return 0;
		}

		__global__ void remap_kernel(const PtrStepSz<uchar3> src, const PtrStep<float> mapx, const PtrStep<float> mapy, PtrStepSz<uchar3> dst)
		{
			const int x = blockDim.x * blockIdx.x + threadIdx.x;
			const int y = blockDim.y * blockIdx.y + threadIdx.y;

			if (x < dst.cols && y < dst.rows)
			{
				float xcoo = mapx.ptr(y)[x];
				float ycoo = mapy.ptr(y)[x];
				int X = trunc(xcoo);
				int Y = trunc(ycoo);
				float xfrac = xcoo - X;
				float yfrac = ycoo - Y;
				if (0 <= X && X < src.cols && 0 <= Y && Y < src.rows)
				{
					//uchar3 p[2][2];
					uchar3 p00 = src(Y, X);
					uchar3 p10 = src(Y + 1, X);
					uchar3 p01 = src(Y, X + 1);
					uchar3 p11 = src(Y + 1, X + 1);
					// bilinear interpolation 
					dst.ptr(y)[x].x = (p00.x * (1 - xfrac) + p01.x * xfrac) * (1 - yfrac) + (p10.x * (1 - xfrac) + p11.x * xfrac) * yfrac;
					dst.ptr(y)[x].y = (p00.y * (1 - xfrac) + p01.y * xfrac) * (1 - yfrac) + (p10.y * (1 - xfrac) + p11.y * xfrac) * yfrac;
					dst.ptr(y)[x].z = (p00.z * (1 - xfrac) + p01.z * xfrac) * (1 - yfrac) + (p10.z * (1 - xfrac) + p11.z * xfrac) * yfrac;
				}
			}
		}

		double remap(const GpuMat src, GpuMat dst, const GpuMat mapx, const GpuMat mapy)
		{
			dim3 threads(16, 16);
			dim3 grid((src.cols) / (threads.x), (src.rows) / (threads.y));

			int64 start = getTickCount();
			remap_kernel << <grid, threads >> >(src, mapx, mapy, dst);
			int64 stop = getTickCount();

			hipDeviceSynchronize();
			return (stop - start) * 1000000 / getTickFrequency();
		}
	}
}
